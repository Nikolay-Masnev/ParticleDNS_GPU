#include "hip/hip_runtime.h"
#include "langevin.h"

#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "helper_functions.h" 

__constant__ double Re = 2500;

__device__ void printArray( uint64_t *v, unsigned long long int size)
{
    for(unsigned long long int i = 0; i < size; ++i)
        printf("%lli\n", v[i]);
}

__global__ void setup_kernel(hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x  + blockIdx.x * blockDim.x;
    hiprand_init(seed, id , 0, &state[id]);
}

__device__ double Sigma(double r)
{
    return 10 * (0.2 * tanh(0.5 * r) - 0.1 * tanh(0.1 * r));
}

__device__ double D(double r, double L)
{
    return 0.01 * sqrt(0.1 + pow(r/L, 2));
}

__device__ double M(double r, double L)
{
    return 1e2 * (1 + 1e4 * pow(r/L, 2) );
}

__device__ double tau_corr(double r, double L)
{
    return 1/sqrt(15 * M(r, L));
}

__global__ void numericalProcedure(unsigned long long int *d_concentration,
    const input_params params, const unsigned long long int size, hiprandState *state,
    float *d_tr_x, float *d_tr_y, float *d_tr_wx, float *d_tr_wy, unsigned long long int tr_points)
{
    unsigned long long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = state[idx];
    hiprand_init(idx, 0, 0, &state[idx]);

    double L = params.BoxSize;
    double a = params.a;
    double r_bin = L / size / 2;
    double sqrt12 = sqrt((float)12);
    double tau_invert = pow(L, 2) / (pow(a,2) * Re);
    double tau = 1/tau_invert;
    double dt = tau / 10;
    double sqrt_dt = sqrt(dt);
    double dt_tau_invert = dt * tau_invert;
    double sqrt_dt_12 = sqrt_dt * sqrt12;
    double dx, dy, w_x, w_y, kx_1, kx_2, ky_1, ky_2, 
    kwx_1, kwx_2, kwy_1, kwy_2;

    dx = 0;
    dy = 0;
    w_x = 0;
    w_y = 0;
    kx_1 = 0;
    kx_2 = 0;
    ky_1 = 0;
    ky_2 = 0;
    kwx_1 = 0;
    kwx_2 = 0;
    kwy_1 = 0;
    kwy_2 = 0;

    unsigned long long int steps = params.numSteps;
    unsigned long long int ind = 0;

    double x = L * (hiprand_uniform(&localState) - 0.5);
    double y = L * (hiprand_uniform(&localState) - 0.5);
    double r = sqrt(x*x + y*y);

    double W1 = 0;
    double W2 = 0;
    double W3 = 0;
    double W4 = 0;
    double W5 = 0;
    double W6 = 0;
    double W7 = 0;
    double W8 = 0;
    double W1_old = 0;
    double W2_old = 0;
    double W3_old = 0;
    double W4_old = 0;
    double W5_old = 0;
    double W6_old = 0;
    double W7_old = 0;
    double W8_old = 0;
    
    double rho = 0;
    double sqrt_one_rho = 0;

    __syncthreads();

    for(unsigned long long int i = 0; i < steps; ++i)
    {   
        // rho = exp(-dt/tau_corr(r, L));
        // sqrt_one_rho = sqrt(1 - rho * rho);
        rho = 0;
        sqrt_one_rho = 1;

        W1 = W1_old * rho + sqrt_one_rho * (hiprand_uniform(&localState) - 0.5);
        W2 = W2_old * rho + sqrt_one_rho * (hiprand_uniform(&localState) - 0.5);
        W3 = W3_old * rho + sqrt_one_rho * (hiprand_uniform(&localState) - 0.5);
        W4 = W4_old * rho + sqrt_one_rho * (hiprand_uniform(&localState) - 0.5);
        W5 = W5_old * rho + sqrt_one_rho * (hiprand_uniform(&localState) - 0.5);
        W6 = W6_old * rho + sqrt_one_rho * (hiprand_uniform(&localState) - 0.5);
       	W7 = W7_old * rho + sqrt_one_rho * (hiprand_uniform(&localState) - 0.5);
	    W8 = W8_old * rho + sqrt_one_rho * (hiprand_uniform(&localState) - 0.5);
        W1_old = W1;
        W2_old = W2;
        W3_old = W3;
        W4_old = W4;
        W5_old = W5;
        W6_old = W6;
        W7_old = W7;
        W8_old = W8;
        
        kx_1 = 0 * dt * w_x + sqrt_dt_12 * W5 * sqrt(D(r, L));
        ky_1 = 0 * dt * w_y + sqrt_dt_12 * W6 * sqrt(D(r, L));
        kwx_1 = - dt_tau_invert * w_x + sqrt_dt_12 * W1 * sqrt(D(r,L));
        kwy_1 = - dt_tau_invert * w_y + sqrt_dt_12 * W2 * sqrt(D(r,L));

        r = sqrt((x+kx_1)*(x+kx_1) + (y+ky_1)*(y+ky_1));

        kx_2 = 0 * dt * (w_x + kwx_1) + sqrt_dt_12 * W7 * sqrt(D(r, L));
        ky_2 = 0 * dt * (w_y + kwy_1) + sqrt_dt_12 * W8 * sqrt(D(r, L));
        kwx_2 = - dt_tau_invert * (w_x + kwx_1) + sqrt_dt_12 * W3 * sqrt(D(r,L));
        kwy_2 = - dt_tau_invert * (w_y + kwy_1) + sqrt_dt_12 * W4 * sqrt(D(r,L));

        dx = 0.5 * (kx_1 + kx_2);
        dy = 0.5 * (ky_1 + ky_2);
        w_x += 0.5 * (kwx_1 + kwx_2);
        w_y += 0.5 * (kwy_1 + kwy_2);

        if(x + dx > L/2)
        {
            x = L - x - dx;
            w_x *= -1;
            W1 *= -1;
            W3 *= -1;
        }
        else if (x + dx < -L/2)
        {
            x = -L  - x - dx;
	        w_x *= -1;
	        W1 *= -1;
            W3 *= -1;
        }
        else
        {
            x += dx;
        }
        
        if(y + dy > L/2)
        {
            y = L - y - dy;
            w_y *= -1;
            W2 *= -1;
            W4 *= -1;
        }
        else if (y + dy < -L/2)
        {
            y = -L  - y - dy;
	        w_y *= -1;
	        W2 *= -1;
            W4 *= -1;
        }
        else
        {
            y += dy;
        }

        r = sqrt(abs(x*x + y*y));

#ifdef CONCENTRATION
        ind = min(int(r / r_bin), int(size-1));
        atomicAdd(&d_concentration[ind], 1);
#endif // CONCENTRATION

#ifdef TRAJECTORY
        if( i < tr_points)
        {
            d_tr_x[i] = x;
            d_tr_y[i] = y;
            d_tr_wx[i] = w_x;
            d_tr_wy[i] = w_y;
        }
#endif // TRAJECTORY
    }

    __syncthreads();
}
