#include "hip/hip_runtime.h"
#include "langevin.h"

#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "helper_functions.h" 

__constant__ double Re = 2500;
#define  M_PI  3.14159265358979323846

__device__ void printArray( uint64_t *v, unsigned long long int size)
{
    for(unsigned long long int i = 0; i < size; ++i)
        printf("%lli\n", v[i]);
}

__global__ void setup_kernel(hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x  + blockIdx.x * blockDim.x;
    hiprand_init(seed, id , 0, &state[id]);
}

__device__ double Sigma(double r)
{
    return 10 * (0.2 * tanh(0.5 * r) - 0.1 * tanh(0.1 * r));
}

__device__ double D(double r, double L)
{
    return (1 + pow(r/L, 2));
}

__device__ double dD_dx(double x, double y, double L)
{
    return 2 * x/ (L * L);
}

__device__ double dD_dy(double x, double y, double L)
{
    return 2 * y/ (L * L);
}

__device__ double K(double r, double L)
{
    return (1 + pow(r/L, 2));
}

__device__ double dK_dx(double x, double y, double L)
{
    return 2 * x/ (L * L);
}

__device__ double dK_dy(double x, double y, double L)
{
    return 2 * y/ (L * L);
}

__device__ double M(double r, double L)
{
    return 1e2 * (1 + 1e4 * pow(r/L, 2) );
}

__device__ double tau_corr(double r, double L)
{
    return 1/sqrt(15 * M(r, L));
}

__global__ void numericalProcedure(unsigned long long int *d_concentration,
    const input_params params, const unsigned long long int size, hiprandState *state,
    float *d_tr_x, float *d_tr_y, float *d_tr_wx, float *d_tr_wy, unsigned long long int tr_points)
{
    unsigned long long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = state[idx];
    hiprand_init(idx, 0, 0, &state[idx]);

    double L = params.BoxSize;
    double a = params.a;
    double r_bin = L / size;
    double sqrt12 = sqrt((float)12);
    double tau_invert = pow(L, 2) / (pow(a,2) * Re) * 100;
    double tau = 1/tau_invert;
    double dt = tau/10;
    double sqrt_dt = sqrt(dt);
    double dt_tau_invert = dt * tau_invert;

    double dx = 0, dy = 0, x = 0, y = 0;
    double dw_x = 0, dw_y = 0, w_x = 0, w_y = 0;

    unsigned long long int steps = params.numSteps;
    unsigned long long int ind = 0;

    double r = L * hiprand_uniform(&localState);
    double phi = 2 * M_PI * hiprand_uniform(&localState);

    x = r * sin(phi);
    y = r * cos(phi);

    double dW1 = 0;
    double dW2 = 0;
    double dW3 = 0;
    double dW4 = 0;

    __syncthreads();

    for(unsigned long long int i = 0; i < steps; ++i)
    {   
        dW1 = hiprand_normal(&localState);
        dW2 = hiprand_normal(&localState);
        dW3 = hiprand_normal(&localState);
        dW4 = hiprand_normal(&localState);

        dx = (w_x + dD_dx(x,y,L)) * dt + sqrt(2 * D(r, L)) * sqrt_dt * dW1;
        dy = (w_y + dD_dy(x,y,L)) * dt + sqrt(2 * D(r, L)) * sqrt_dt * dW2;

        dw_x = (-tau_invert * w_x + dK_dx(x, y, L)) * dt + sqrt(2 * K(r, L)) * sqrt_dt * dW3;
        dw_y = (-tau_invert * w_y + dK_dy(x, y, L)) * dt + sqrt(2 * K(r, L)) * sqrt_dt * dW4;

        x += dx;
        y += dy;

        if(w_x * (w_x + dw_x) < 0)
            w_x = 0;
        else
            w_x += dw_x;

        if(w_y * (w_y + dw_y) < 0)
            w_y = 0;
        else
            w_y += dw_y;

        r = sqrt(x*x + y*y);

        if(r > L)
        {
            x -= dx;
            y -= dy;

            if(w_x * x + w_y * y > 0)
            {
                w_x = (x/r) * w_x + (y/r) * w_y;
                w_y = -(y/r) * w_x + (x/r) * w_y;
            }
        }

        r = sqrt(x*x + y*y);

#ifdef CONCENTRATION
        ind = min(int(r / r_bin), int(size-1));
        atomicAdd(&d_concentration[ind], 1);

	    if(int(r/r_bin) > size)
        {
            printf("r = %f\n", r);
        }
#endif // CONCENTRATION

#ifdef TRAJECTORY
        if( i < tr_points)
        {
            d_tr_x[i] = x;
            d_tr_y[i] = y;
            d_tr_wx[i] = w_x;
            d_tr_wy[i] = w_y;
        }
#endif // TRAJECTORY
    }

    __syncthreads();
}
