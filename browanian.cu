#include "hip/hip_runtime.h"
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "helper_functions.h" 

#include "dataio.h"
#include "input_params.h"
#include "langevin.h"

#define THREADS 1000
#define BLOCKS  100

unsigned long long int nBins = 100;
unsigned long long int tr_points = 1e4;

template <typename T>
void printArray(T* arr, unsigned long long int size)
{
    for(unsigned long long int i = 0; i < size; ++i)
    {
        std::cout << arr[i] << ' '; 
    }

    std::cout << '\n';
}

void normalizeArray(float* arr, unsigned long long int* uint_arr, unsigned long long int size)
{
    unsigned long long int sum = 0;

    for(unsigned long long int i = 0; i < size; ++i)
    {
        sum+=uint_arr[i];
    }

    for(unsigned long long int i = 0; i < size; ++i)
    {
        arr[i] =float(uint_arr[i]) / sum;
    }
}

void normalizeVariance(double* arr, unsigned long long int* uint_arr, unsigned long long int size)
{
    printArray(arr, size);
    printArray(uint_arr, size);

    for(unsigned long long int i = 0; i < size; ++i)
    {
        arr[i] = arr[i] / (uint_arr[i]+1);
	arr[i] = std::sqrt(arr[i]);
    }
}

int main(int argc, char *argv[])
{ 
    // reading params
    printf("reading params\n");
    std::string paramsPath(argv[1]);
    input_params data;
    readParams(data, paramsPath);
    printParams(data);

    // allocate HOST memory
    printf("allocate HOST memory\n");

    float *concentration = nullptr;
    float *concentration_2D = nullptr;
    unsigned long long int *uint64_t_concentration = nullptr;
    unsigned long long int *uint64_t_concentration_2D = nullptr;
    float *tr_x = nullptr;
    float *tr_y = nullptr;
    float *tr_wx = nullptr;
    float *tr_wy = nullptr;
    double *velocity_variance = nullptr;
    unsigned long long int *variance_counter = nullptr;

#ifdef CONCENTRATION
    checkCudaErrors(hipHostMalloc((void **)&concentration, nBins * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&uint64_t_concentration, nBins * sizeof(unsigned long long int)));
    memset(concentration, 0, nBins * sizeof(float));
    memset(uint64_t_concentration, 0, nBins * sizeof(unsigned long long int));
#endif /* CONCENTRATION */

#ifdef _2D_HISTOGRAM
    checkCudaErrors(hipHostMalloc((void **)&concentration_2D, nBins * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&uint64_t_concentration_2D, 4 * nBins * nBins * sizeof(unsigned long long int)));
    memset(concentration_2D, 0, 4 * nBins * nBins * sizeof(float));
    memset(uint64_t_concentration_2D, 0, 4 * nBins * nBins * sizeof(unsigned long long int));
#endif /* 2D_HISTOGRAM */

#ifdef TRAJECTORY
    uint64_t tr_nbytes = tr_points * sizeof(float);
    checkCudaErrors(hipHostMalloc((void **)&tr_x, tr_nbytes ));
    checkCudaErrors(hipHostMalloc((void **)&tr_y, tr_nbytes ));
    checkCudaErrors(hipHostMalloc((void **)&tr_wx, tr_nbytes ));
    checkCudaErrors(hipHostMalloc((void **)&tr_wy, tr_nbytes ));
    memset(tr_x, 0, nbytes);
    memset(tr_y, 0, nbytes);
    memset(tr_wx, 0, nbytes);
    memset(tr_wy, 0, nbytes);
#endif /* TRAJECTORY */

#ifdef VELOCITY_VARIANCE
    checkCudaErrors(hipHostMalloc((void **)&velocity_variance, nBins * sizeof(double)));
    checkCudaErrors(hipHostMalloc((void **)&variance_counter, nBins * sizeof(unsigned long long int)));
    memset(velocity_variance, 0, nBins * sizeof(double));
    memset(variance_counter, 0, nBins * sizeof(unsigned long long int));
#endif /* VELOCITY_VARIANCE */
    
    // allocate DEVISE memory
    printf("allocate DEVISE memory\n");

    unsigned long long int *d_concentration = nullptr;
    unsigned long long int *d_concentration_2D = nullptr;
    float *d_tr_x = nullptr;
    float *d_tr_y = nullptr;
    float *d_tr_wx = nullptr;
    float *d_tr_wy = nullptr;
    double *d_velocity_variance = nullptr;
    unsigned long long int *d_variance_counter = nullptr;

#ifdef CONCENTRATION
    checkCudaErrors(hipMalloc((void **)&d_concentration, nBins * sizeof(unsigned long long int)));
    hipMemset(d_concentration, 0, nBins * sizeof(unsigned long long int));
#endif /* CONCENTRATION */

#ifdef _2D_HISTOGRAM
    checkCudaErrors(hipMalloc((void **)&d_concentration_2D, 4 * nBins * nBins * sizeof(unsigned long long int)));
    hipMemset(d_concentration_2D, 0, 4 * nBins * nBins * sizeof(unsigned long long int));
#endif /* 2D_HISTOGRAM */

#ifdef TRAJECTORY
    checkCudaErrors(hipMalloc((void **)&d_tr_x, tr_nbytes));
    checkCudaErrors(hipMalloc((void **)&d_tr_y, tr_nbytes));
    checkCudaErrors(hipMalloc((void **)&d_tr_wx, tr_nbytes));
    checkCudaErrors(hipMalloc((void **)&d_tr_wy, tr_nbytes));
    
    hipMemset(d_tr_x, 0, tr_nbytes);
    hipMemset(d_tr_y, 0, tr_nbytes);
    hipMemset(d_tr_wx, 0, tr_nbytes);
    hipMemset(d_tr_wy, 0, tr_nbytes);
#endif /* TRAJECTORY */

#ifdef VELOCITY_VARIANCE
    checkCudaErrors(hipMalloc((void **)&d_velocity_variance, nBins * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_variance_counter, nBins * sizeof(unsigned long long int)));
    hipMemset(d_velocity_variance, 0, nBins * sizeof(double));
    hipMemset(d_variance_counter, 0, nBins * sizeof(unsigned long long int));
#endif /* VELOCITY_VARIANCE */

    // create cuda event handles
    printf("create cuda event handles\n");
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    checkCudaErrors(hipDeviceSynchronize());

    // set random number generator
    printf("set random number generator\n");
    hiprandState *devState;
    checkCudaErrors(hipMalloc((void**)&devState, THREADS * BLOCKS * sizeof(hiprandState)));
    time_t t;
    time(&t);
    setup_kernel<<<THREADS, BLOCKS, 0, 0>>>(devState, (unsigned long) t);

    // copy data from host to devise
    printf("copy data from host to devise\n");

#ifdef CONCENTRATION
    hipMemcpy(d_concentration, uint64_t_concentration, nBins * sizeof(unsigned long long int), hipMemcpyHostToDevice);
#endif /* CONCENTRATION */

#ifdef _2D_HISTOGRAM
    hipMemcpy(d_concentration_2D, uint64_t_concentration_2D, 4 * nBins * nBins * sizeof(unsigned long long int), hipMemcpyHostToDevice);
#endif /* 2D_HISTOGRAM */

#ifdef TRAJECTORY
    hipMemcpy(d_tr_x, tr_x, tr_nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_tr_y, tr_y, tr_nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_tr_wx, tr_wx, tr_nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_tr_wy, tr_wy, tr_nbytes, hipMemcpyHostToDevice);
#endif /* TRAJECTORY */

#ifdef VELOCITY_VARIANCE
    hipMemcpy(d_velocity_variance, velocity_variance, nBins * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_variance_counter, variance_counter, nBins * sizeof(unsigned long long int), hipMemcpyHostToDevice);
#endif /* VELOCITY_VARIANCE */

    //start
    printf("start\n");
    float gpu_time = 0.0f;
    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    // main loop
    printf("main loop\n");

    int nBins_2D =  4 * nBins * nBins;
    printf("Launch kernel\n");   
    numericalProcedure<<<THREADS, BLOCKS, 0, 0>>>(d_concentration, data,  nBins, devState, 
        d_tr_x, d_tr_y, d_tr_wx, d_tr_wy, tr_points, d_concentration_2D, nBins_2D, d_velocity_variance, d_variance_counter, nBins);
    checkCudaErrors(hipDeviceSynchronize());

#ifdef CONCENTRATION
    hipMemcpy(uint64_t_concentration, d_concentration, nBins * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
#endif /* CONCENTRATION */
    
#ifdef TRAJECTORY
    hipMemcpy(tr_x, d_tr_x, tr_nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(tr_y, d_tr_y, tr_nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(tr_wx, d_tr_wx, tr_nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(tr_wy, d_tr_wy, tr_nbytes, hipMemcpyDeviceToHost);
#endif /* TRAJECTORY */

#ifdef _2D_HISTOGRAM
    hipMemcpy(uint64_t_concentration_2D, d_concentration_2D, 4 * nBins * nBins * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
#endif /* 2D_HISTOGRAM */

#ifdef VELOCITY_VARIANCE
    hipMemcpy(velocity_variance, d_velocity_variance, nBins * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(variance_counter, d_variance_counter, nBins * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
#endif /* VELOCITY_VARIANCE */

    // stop
    printf("stop\n");
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    checkCudaErrors(hipDeviceSynchronize());

    // normalize
#ifdef CONCENTRATION 
    normalizeArray(concentration, uint64_t_concentration,nBins);
#endif /* CONCENTRATION */

#ifdef _2D_HISTOGRAM
    normalizeArray(concentration_2D, uint64_t_concentration_2D, 4 * nBins * nBins);
#endif /* 2D_HISTOGRAM */

#ifdef VELOCITY_VARIANCE
    normalizeVariance(velocity_variance, variance_counter, nBins);
#endif /* VELOCITY_VARIANCE */

    // save distribution
    printf("save dist\n");

#ifdef CONCENTRATION  
    saveHist(concentration, argv[2], nBins);
#endif /* CONCENTRATION3 */

#ifdef TRAJECTORY   
    saveHist(tr_x, argv[3], tr_points);
    saveHist(tr_y, argv[4], tr_points);
    saveHist(tr_wx, argv[5], tr_points);
    saveHist(tr_wy, argv[6], tr_points);
#endif /* TRAJECTORY */

#ifdef _2D_HISTOGRAM
    saveHist(concentration_2D, argv[7], 4 * nBins * nBins);
#endif /* 2D_HISTOGRAM */

#ifdef VELOCITY_VARIANCE
    saveHist(velocity_variance, argv[8], nBins);
#endif /* VELOCITY_VARIANCE */

    // free memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipFree(devState));

#ifdef CONCENTRATION  
    checkCudaErrors(hipHostFree(concentration));
    checkCudaErrors(hipHostFree(uint64_t_concentration));
    checkCudaErrors(hipFree(d_concentration));
#endif /* CONCENTRATION */

#ifdef _2D_HISTOGRAM
    checkCudaErrors(hipHostFree(concentration_2D));
    checkCudaErrors(hipHostFree(uint64_t_concentration_2D));
    checkCudaErrors(hipFree(d_concentration_2D));
#endif /* 2D_HISTOGRAM */

#ifdef TRAJECTORY
    checkCudaErrors(hipHostFree(tr_x));
    checkCudaErrors(hipHostFree(tr_y));
    checkCudaErrors(hipHostFree(tr_wx));
    checkCudaErrors(hipHostFree(tr_wy));
    checkCudaErrors(hipFree(d_tr_x));
    checkCudaErrors(hipFree(d_tr_y));
    checkCudaErrors(hipFree(d_tr_wx));
    checkCudaErrors(hipFree(d_tr_wy));
#endif /* TRAJECTORY */

#ifdef VELOCITY_VARIANCE
    checkCudaErrors(hipHostFree(velocity_variance));
    checkCudaErrors(hipHostFree(variance_counter));
    checkCudaErrors(hipFree(d_velocity_variance));
    checkCudaErrors(hipFree(d_variance_counter));
#endif /* VELOCITY_VARIANCE */

    return EXIT_SUCCESS;
}
