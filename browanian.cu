#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "helper_functions.h" 

#include "dataio.h"
#include "input_params.h"
#include "langevin.h"

#define THREADS 1000
#define BLOCKS  1

uint64_t nBins = 100;

void printArray(float* arr, uint64_t size)
{
    for(uint64_t i = 0; i < size; ++i)
    {
        printf("%lli: %f\n", i, arr[i]);
    }
}

void normalizeArray(float* arr, uint64_t size)
{
    float sum = 0;

    for(uint64_t i = 0; i < size; ++i)
    {
        sum+=arr[i];
    }

    for(uint64_t i = 0; i < size; ++i)
    {
        arr[i] /= sum;
    }
}

int main(int argc, char *argv[])
{ 
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    // reading params
    printf("reading params\n");
    std::string paramsPath(argv[1]);
    input_params data;
    readParams(data, paramsPath);

    // allocate HOST memory
    printf("allocate HOST memory\n");
    uint64_t nbytes = nBins * sizeof(float);

    float *concentration = 0;

    checkCudaErrors(hipHostMalloc((void **)&concentration, nbytes));

    memset(concentration, 0, nbytes);

    // allocate DEVISE memory
    printf("allocate DEVISE memory\n");
    float *d_concentration = 0;

    checkCudaErrors(hipMalloc((void **)&d_concentration, nbytes));

    hipMemset(d_concentration, 0, nbytes);

    // create cuda event handles
    printf("create cuda event handles\n");
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    checkCudaErrors(hipDeviceSynchronize());

    // set random number generator
    printf("set random number generator\n");
    hiprandState *devState;
    checkCudaErrors(hipMalloc((void**)&devState, THREADS * BLOCKS * sizeof(hiprandState)));
    time_t t;
    time(&t);
    setup_kernel<<<THREADS, BLOCKS, 0, 0>>>(devState, (unsigned long) t);

    // copy data from host to devise
    printf("copy data from host to devise\n");
    hipMemcpy(d_concentration, concentration, nbytes, hipMemcpyHostToDevice);

    //start
    printf("start\n");
    float gpu_time = 0.0f;
    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    // main loop
    printf("main loop\n");
    numericalProcedure<<<THREADS, BLOCKS, 0, 0>>>(d_concentration, data,  nBins, devState);
    checkCudaErrors(hipDeviceSynchronize());

    hipMemcpy(concentration, d_concentration, nbytes, hipMemcpyDeviceToHost);

    // stop
    printf("stop\n");
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    printf("time spent executing by the GPU: %.2f\n", gpu_time);

    checkCudaErrors(hipDeviceSynchronize());

    // normalize
    uint64_t sum = 0;

    for(uint64_t i = 0; i < uint64_t(nBins); ++i)
    {
        sum+= uint64_t(concentration[i]);
    }

    printf("sum = %lli\n", sum);

    normalizeArray(concentration, nBins);

    // save distribution
    printf("save dist\n");
    saveHist(concentration, argv[2], nBins);

    // free memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(concentration));
    checkCudaErrors(hipFree(d_concentration));
    checkCudaErrors(hipFree(devState));

    return EXIT_SUCCESS;
}