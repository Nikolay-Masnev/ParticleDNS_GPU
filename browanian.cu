#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include <math.h>
#include <random>
#include <fstream>
#include <sstream>
#include <iterator>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "helper_functions.h" 

#include "dataio.h"
#include "input_params.h"
#include "langevin.h"

#define THREADS 10
#define BLOCKS  1

uint64_t nBins = 100;
uint64_t autocorr_nBins = 1000;

void normalizeArray(double* arr, uint64_t size)
{
    double sum = 0;

    for(uint64_t i = 0; i < size; ++i)
    {
        sum+=arr[i];
    }

    for(uint64_t i = 0; i < size; ++i)
    {
        arr[i] /= sum;
    }
}

int main(int argc, char *argv[])
{ 
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    // reading params
    std::string paramsPath(argv[1]);
    input_params data;
    readParams(data, paramsPath);

    // allocate HOST memory
    uint64_t nbytes = nBins * sizeof(double);
    uint64_t autocorr_nbytes = autocorr_nBins * sizeof(double);

    double *concentration = 0;
    double *velocityVariance = 0;
    double *angleVariance = 0;
    double *pdf_vel = 0;
    double *w_autocorrelator = 0;
    double *phi_autocorrelator = 0;

    checkCudaErrors(hipHostMalloc((void **)&concentration, nbytes));
    checkCudaErrors(hipHostMalloc((void **)&velocityVariance, nbytes));
    checkCudaErrors(hipHostMalloc((void **)&angleVariance, nbytes));
    checkCudaErrors(hipHostMalloc((void **)&pdf_vel, nbytes));
    checkCudaErrors(hipHostMalloc((void **)&w_autocorrelator, autocorr_nbytes));
    checkCudaErrors(hipHostMalloc((void **)&phi_autocorrelator, autocorr_nbytes));

    memset(concentration, 0, nbytes);
    memset(velocityVariance, 0, nbytes);
    memset(angleVariance, 0, nbytes);
    memset(pdf_vel, 0, nbytes);
    memset(w_autocorrelator, 0, autocorr_nbytes);
    memset(phi_autocorrelator, 0, autocorr_nbytes);

    // allocate DEVISE memory
    double *d_concentration = 0;
    double *d_velocityVariance = 0;
    double *d_angleVariance = 0;
    double *d_pdf_vel = 0;
    double *d_w_autocorrelator = 0;
    double *d_phi_autocorrelator = 0;

    checkCudaErrors(hipMalloc((void **)&d_concentration, nbytes));
    checkCudaErrors(hipMalloc((void **)&d_velocityVariance, nbytes));
    checkCudaErrors(hipMalloc((void **)&d_angleVariance, nbytes));
    checkCudaErrors(hipMalloc((void **)&d_pdf_vel, nbytes));
    checkCudaErrors(hipMalloc((void **)&d_w_autocorrelator, autocorr_nbytes));
    checkCudaErrors(hipMalloc((void **)&d_phi_autocorrelator, autocorr_nbytes));

    memset(d_concentration, 0, nbytes);
    memset(d_velocityVariance, 0, nbytes);
    memset(d_angleVariance, 0, nbytes);
    memset(d_pdf_vel, 0, nbytes);
    memset(d_w_autocorrelator, 0, autocorr_nbytes);
    memset(d_phi_autocorrelator, 0, autocorr_nbytes);

    //set kernel launch configuration
    dim3 threads = dim3(THREADS, 1, 1);
    dim3 blocks = dim3(BLOCKS, 1, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    checkCudaErrors(hipDeviceSynchronize());

    // set random number generator
    hiprandState *devState;
    checkCudaErrors(hipMalloc((void**)&devState, THREADS * BLOCKS * sizeof(hiprandState)));
    time_t t;
    time(&t);
    setup_kernel<<<THREADS, BLOCKS, 0, 0>>>(devState, (unsigned long) t);

    // copy data from host to devise
    hipMemcpy(d_concentration, concentration, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_velocityVariance, velocityVariance, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_angleVariance, angleVariance, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_pdf_vel, pdf_vel, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_w_autocorrelator, w_autocorrelator, autocorr_nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_phi_autocorrelator, phi_autocorrelator, autocorr_nbytes, hipMemcpyHostToDevice);

    //start
    float gpu_time = 0.0f;
    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    // main loop
    numericalProcedure<<<THREADS, BLOCKS, 0, 0>>>(d_concentration, d_velocityVariance, d_pdf_vel, 
    d_w_autocorrelator, d_phi_autocorrelator, data,  nBins,  autocorr_nBins , devState);
    checkCudaErrors(hipDeviceSynchronize());

    hipMemcpy(concentration, d_concentration, nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(velocityVariance, d_velocityVariance, nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(pdf_vel, d_pdf_vel, nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(w_autocorrelator, d_w_autocorrelator, autocorr_nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(phi_autocorrelator, d_phi_autocorrelator, autocorr_nbytes, hipMemcpyDeviceToHost);

    // stop
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    printf("time spent executing by the GPU: %.2f\n", gpu_time);

    // norm pdf
    normalizeArray(concentration, nBins);
    normalizeArray(velocityVariance, nBins);
    normalizeArray(pdf_vel, nBins);

    normalizeArray(w_autocorrelator, autocorr_nBins);
    normalizeArray(phi_autocorrelator, autocorr_nBins);

    // save distribution
    saveHist(concentration, argv[2], nBins);
    saveHist(velocityVariance, argv[3], nBins);
    saveHist(pdf_vel, argv[4], nBins);

    saveHist(w_autocorrelator, argv[5], autocorr_nBins);
    saveHist(phi_autocorrelator, argv[6], autocorr_nBins);

    // free memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(concentration));
    checkCudaErrors(hipHostFree(velocityVariance));
    checkCudaErrors(hipHostFree(pdf_vel));
    checkCudaErrors(hipHostFree(w_autocorrelator));
    checkCudaErrors(hipHostFree(phi_autocorrelator));

    checkCudaErrors(hipFree(d_concentration));
    checkCudaErrors(hipFree(d_velocityVariance));
    checkCudaErrors(hipFree(d_pdf_vel));
    checkCudaErrors(hipFree(d_w_autocorrelator));
    checkCudaErrors(hipFree(d_phi_autocorrelator));

    checkCudaErrors(hipFree(devState));

    return EXIT_SUCCESS;
}