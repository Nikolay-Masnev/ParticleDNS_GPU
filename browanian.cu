#include "hip/hip_runtime.h"
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "helper_functions.h" 

#include "dataio.h"
#include "input_params.h"
#include "langevin.h"

#define THREADS 100
#define BLOCKS  1

uint64_t nBins = 100;
uint64_t autocorr_nBins = 1000;

void normalizeArray(double* arr, uint64_t size)
{
    double sum = 0;

    for(uint64_t i = 0; i < size; ++i)
    {
        sum+=arr[i];
    }

    for(uint64_t i = 0; i < size; ++i)
    {
        arr[i] /= sum;
    }
}

int main(int argc, char *argv[])
{ 
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    // reading params
    printf("reading params\n");
    std::string paramsPath(argv[1]);
    input_params data;
    readParams(data, paramsPath);

    // allocate HOST memory
    printf("allocate HOST memory\n");
    uint64_t nbytes = nBins * sizeof(double);
    uint64_t autocorr_nbytes = autocorr_nBins * sizeof(double);

    double *concentration = 0;
    double *velocityVariance = 0;
    double *angleVariance = 0;
    double *pdf_vel = 0;
    double *w_autocorrelator = 0;
    double *phi_autocorrelator = 0;

    checkCudaErrors(hipHostMalloc((void **)&concentration, nbytes));
    checkCudaErrors(hipHostMalloc((void **)&velocityVariance, nbytes));
    checkCudaErrors(hipHostMalloc((void **)&angleVariance, nbytes));
    checkCudaErrors(hipHostMalloc((void **)&pdf_vel, nbytes));
    checkCudaErrors(hipHostMalloc((void **)&w_autocorrelator, autocorr_nbytes));
    checkCudaErrors(hipHostMalloc((void **)&phi_autocorrelator, autocorr_nbytes));

    memset(concentration, 0, nbytes);
    memset(velocityVariance, 0, nbytes);
    memset(angleVariance, 0, nbytes);
    memset(pdf_vel, 0, nbytes);
    memset(w_autocorrelator, 0, autocorr_nbytes);
    memset(phi_autocorrelator, 0, autocorr_nbytes);

    // allocate DEVISE memory
    printf("allocate DEVISE memory\n");
    double *d_concentration = 0;
    double *d_velocityVariance = 0;
    double *d_angleVariance = 0;
    double *d_pdf_vel = 0;
    double *d_w_autocorrelator = 0;
    double *d_phi_autocorrelator = 0;

    checkCudaErrors(hipMalloc((void **)&d_concentration, nbytes));
    checkCudaErrors(hipMalloc((void **)&d_velocityVariance, nbytes));
    checkCudaErrors(hipMalloc((void **)&d_angleVariance, nbytes));
    checkCudaErrors(hipMalloc((void **)&d_pdf_vel, nbytes));
    checkCudaErrors(hipMalloc((void **)&d_w_autocorrelator, autocorr_nbytes));
    checkCudaErrors(hipMalloc((void **)&d_phi_autocorrelator, autocorr_nbytes));

    hipMemset(d_concentration, 0, nbytes);
    hipMemset(d_velocityVariance, 0, nbytes);
    hipMemset(d_angleVariance, 0, nbytes);
    hipMemset(d_pdf_vel, 0, nbytes);
    hipMemset(d_w_autocorrelator, 0, autocorr_nbytes);
    hipMemset(d_phi_autocorrelator, 0, autocorr_nbytes);

    //set kernel launch configuration
    printf("set kernel launch configuration\n");
    dim3 threads = dim3(THREADS, 1, 1);
    dim3 blocks = dim3(BLOCKS, 1, 1);

    // create cuda event handles
    printf("create cuda event handles\n");
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    checkCudaErrors(hipDeviceSynchronize());

    // set random number generator
    printf("set random number generator\n");
    hiprandState *devState;
    checkCudaErrors(hipMalloc((void**)&devState, THREADS * BLOCKS * sizeof(hiprandState)));
    time_t t;
    time(&t);
    setup_kernel<<<THREADS, BLOCKS, 0, 0>>>(devState, (unsigned long) t);

    // copy data from host to devise
    printf("copy data from host to devise\n");
    hipMemcpy(d_concentration, concentration, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_velocityVariance, velocityVariance, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_angleVariance, angleVariance, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_pdf_vel, pdf_vel, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_w_autocorrelator, w_autocorrelator, autocorr_nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_phi_autocorrelator, phi_autocorrelator, autocorr_nbytes, hipMemcpyHostToDevice);

    //start
    printf("start\n");
    float gpu_time = 0.0f;
    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    // main loop
    printf("main loop\n");
    numericalProcedure<<<THREADS, BLOCKS, 0, 0>>>(d_concentration, d_velocityVariance, d_pdf_vel, 
    d_w_autocorrelator, d_phi_autocorrelator, data,  nBins,  autocorr_nBins , devState);
    checkCudaErrors(hipDeviceSynchronize());

    hipMemcpy(concentration, d_concentration, nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(velocityVariance, d_velocityVariance, nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(pdf_vel, d_pdf_vel, nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(w_autocorrelator, d_w_autocorrelator, autocorr_nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(phi_autocorrelator, d_phi_autocorrelator, autocorr_nbytes, hipMemcpyDeviceToHost);

    // stop
    printf("stop\n");
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    printf("time spent executing by the GPU: %.2f\n", gpu_time);

    // norm pdf
    printf("norm pdf\n");
    normalizeArray(concentration, nBins);
    //normalizeArray(velocityVariance, nBins);
    //normalizeArray(pdf_vel, nBins);

    //normalizeArray(w_autocorrelator, autocorr_nBins);
    //normalizeArray(phi_autocorrelator, autocorr_nBins);

    // save distribution
    printf("save dist\n");
    saveHist(concentration, argv[2], nBins);
    //saveHist(velocityVariance, argv[3], nBins);
    //saveHist(pdf_vel, argv[4], nBins);

    //saveHist(w_autocorrelator, argv[5], autocorr_nBins);
    //saveHist(phi_autocorrelator, argv[6], autocorr_nBins);

    // free memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(concentration));
    checkCudaErrors(hipHostFree(velocityVariance));
    checkCudaErrors(hipHostFree(pdf_vel));
    checkCudaErrors(hipHostFree(w_autocorrelator));
    checkCudaErrors(hipHostFree(phi_autocorrelator));

    checkCudaErrors(hipFree(d_concentration));
    checkCudaErrors(hipFree(d_velocityVariance));
    checkCudaErrors(hipFree(d_pdf_vel));
    checkCudaErrors(hipFree(d_w_autocorrelator));
    checkCudaErrors(hipFree(d_phi_autocorrelator));

    checkCudaErrors(hipFree(devState));

    return EXIT_SUCCESS;
}