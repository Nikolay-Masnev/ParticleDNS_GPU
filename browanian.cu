#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "helper_functions.h" 

#include "dataio.h"
#include "input_params.h"
#include "langevin.h"

#define THREADS 1000
#define BLOCKS  1

uint64_t nBins = 100;

void printArray(float* arr, uint64_t size)
{
    for(uint64_t i = 0; i < size; ++i)
    {
        printf("%lli: %f\n", i, arr[i]);
    }
}

void normalizeArray(float* arr, uint64_t* uint_arr, uint64_t size)
{
    uint64_t sum = 0;

    for(uint64_t i = 0; i < size; ++i)
    {
        sum+=uint_arr[i];
    }

    for(uint64_t i = 0; i < size; ++i)
    {
        arr[i] =float(uint_arr[i]) / sum;
    }
}

int main(int argc, char *argv[])
{ 
    // reading params
    printf("reading params\n");
    std::string paramsPath(argv[1]);
    input_params data;
    readParams(data, paramsPath);

    // allocate HOST memory
    printf("allocate HOST memory\n");
    uint64_t nbytes = nBins * sizeof(float);    
    float *concentration = 0;
    checkCudaErrors(hipHostMalloc((void **)&concentration, nbytes));
    memset(concentration, 0, nbytes);

    uint64_t *uint64_t_concentration = 0;
    checkCudaErrors(hipHostMalloc((void **)&uint64_t_concentration, nBins * sizeof(uint64_t)));
    memset(concentration, 0, nBins * sizeof(uint64_t));

    // allocate DEVISE memory
    printf("allocate DEVISE memory\n");
    uint64_t *d_concentration = 0;
    checkCudaErrors(hipMalloc((void **)&d_concentration, nBins * sizeof(uint64_t)));
    hipMemset(d_concentration, 0, nBins * sizeof(uint64_t));

    // create cuda event handles
    printf("create cuda event handles\n");
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    checkCudaErrors(hipDeviceSynchronize());

    // set random number generator
    printf("set random number generator\n");
    hiprandState *devState;
    checkCudaErrors(hipMalloc((void**)&devState, THREADS * BLOCKS * sizeof(hiprandState)));
    time_t t;
    time(&t);
    setup_kernel<<<THREADS, BLOCKS, 0, 0>>>(devState, (unsigned long) t);

    // copy data from host to devise
    printf("copy data from host to devise\n");
    hipMemcpy(d_concentration, uint64_t_concentration, nBins * sizeof(uint64_t), hipMemcpyHostToDevice);

    //start
    printf("start\n");
    float gpu_time = 0.0f;
    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    // main loop
    printf("main loop\n");
    numericalProcedure<<<THREADS, BLOCKS, 0, 0>>>(d_concentration, data,  nBins, devState);
    checkCudaErrors(hipDeviceSynchronize());
    hipMemcpy(uint64_t_concentration, d_concentration, nBins * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // stop
    printf("stop\n");
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    printf("time spent executing by the GPU: %.2f\n", gpu_time);

    checkCudaErrors(hipDeviceSynchronize());

    // normalize
    uint64_t sum = 0;

    for(int i = 0; i < nBins; ++i)
    {
        sum += uint64_t_concentration[i];
    }

    printf("sum = %lli\n", sum);

    normalizeArray(concentration, uint64_t_concentration,nBins);

    // save distribution
    printf("save dist\n");
    saveHist(concentration, argv[2], nBins);

    // free memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(concentration));
    checkCudaErrors(hipHostFree(uint64_t_concentration));
    checkCudaErrors(hipFree(d_concentration));
    checkCudaErrors(hipFree(devState));

    return EXIT_SUCCESS;
}