#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "helper_functions.h" 

#include "dataio.h"
#include "input_params.h"
#include "langevin.h"

#define THREADS 1
#define BLOCKS  1

unsigned long long int nBins = 1000;
unsigned long long int tr_points = 1e4;

void printArray(float* arr, unsigned long long int size)
{
    for(unsigned long long int i = 0; i < size; ++i)
    {
        printf("%lli: %f\n", i, arr[i]);
    }
}

void normalizeArray(float* arr, unsigned long long int* uint_arr, unsigned long long int size)
{
    unsigned long long int sum = 0;

    for(unsigned long long int i = 0; i < size; ++i)
    {
        sum+=uint_arr[i];
    }

    for(unsigned long long int i = 0; i < size; ++i)
    {
        arr[i] =float(uint_arr[i]) / sum;
    }
}

int main(int argc, char *argv[])
{ 
    // reading params
    printf("reading params\n");
    std::string paramsPath(argv[1]);
    input_params data;
    readParams(data, paramsPath);

    // allocate HOST memory
    printf("allocate HOST memory\n");
    uint64_t nbytes = nBins * sizeof(float);
    float *concentration = 0;
    checkCudaErrors(hipHostMalloc((void **)&concentration, nbytes));
    memset(concentration, 0, nbytes);

    unsigned long long int *uint64_t_concentration = 0;
    checkCudaErrors(hipHostMalloc((void **)&uint64_t_concentration, nBins * sizeof(unsigned long long int)));
    memset(concentration, 0, nBins * sizeof(unsigned long long int));
    
   
#ifdef TRAJECTORY
    uint64_t tr_nbytes = tr_points * sizeof(float);
    float *tr_x = 0;
    float *tr_y = 0;
    float *tr_wx = 0;
    float *tr_wy = 0;
    checkCudaErrors(hipHostMalloc((void **)&tr_x, tr_nbytes ));
    checkCudaErrors(hipHostMalloc((void **)&tr_y, tr_nbytes ));
    checkCudaErrors(hipHostMalloc((void **)&tr_wx, tr_nbytes ));
    checkCudaErrors(hipHostMalloc((void **)&tr_wy, tr_nbytes ));
    memset(tr_x, 0, nbytes);
    memset(tr_y, 0, nbytes);
    memset(tr_wx, 0, nbytes);
    memset(tr_wy, 0, nbytes);
#endif /* TRAJECTORY */
    
    // allocate DEVISE memory
    printf("allocate DEVISE memory\n");
    unsigned long long int *d_concentration = 0;
    checkCudaErrors(hipMalloc((void **)&d_concentration, nBins * sizeof(unsigned long long int)));
    hipMemset(d_concentration, 0, nBins * sizeof(unsigned long long int));
    
#ifdef TRAJECTORY
    float *d_tr_x = 0;
    float *d_tr_y = 0;
    float *d_tr_wx = 0;
    float *d_tr_wy = 0;
    
    checkCudaErrors(hipMalloc((void **)&d_tr_x, tr_nbytes));
    checkCudaErrors(hipMalloc((void **)&d_tr_y, tr_nbytes));
    checkCudaErrors(hipMalloc((void **)&d_tr_wx, tr_nbytes));
    checkCudaErrors(hipMalloc((void **)&d_tr_wy, tr_nbytes));
    
    hipMemset(d_tr_x, 0, tr_nbytes);
    hipMemset(d_tr_y, 0, tr_nbytes);
    hipMemset(d_tr_wx, 0, tr_nbytes);
    hipMemset(d_tr_wy, 0, tr_nbytes);
#endif /* TRAJECTORY */

    // create cuda event handles
    printf("create cuda event handles\n");
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    checkCudaErrors(hipDeviceSynchronize());

    // set random number generator
    printf("set random number generator\n");
    hiprandState *devState;
    checkCudaErrors(hipMalloc((void**)&devState, THREADS * BLOCKS * sizeof(hiprandState)));
    time_t t;
    time(&t);
    setup_kernel<<<THREADS, BLOCKS, 0, 0>>>(devState, (unsigned long) t);

    // copy data from host to devise
    printf("copy data from host to devise\n");
    hipMemcpy(d_concentration, uint64_t_concentration, nBins * sizeof(unsigned long long int), hipMemcpyHostToDevice);

    //start
    printf("start\n");
    float gpu_time = 0.0f;
    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    // main loop
    printf("main loop\n");
    
#ifdef TRAJECTORY
    numericalProcedure<<<THREADS, BLOCKS, 0, 0>>>(d_concentration, data,  nBins, devState, d_tr_x, d_tr_y, d_tr_wx, d_tr_wy, tr_points);
#else 
    numericalProcedure<<<THREADS, BLOCKS, 0, 0>>>(d_concentration, data,  nBins, devState, NULL, NULL, NULL, NULL, NULL);
#endif /* TRAJECTORY */

    checkCudaErrors(hipDeviceSynchronize());
    hipMemcpy(uint64_t_concentration, d_concentration, nBins * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    
#ifdef TRAJECTORY
    hipMemcpy(tr_x, d_tr_x, tr_nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(tr_y, d_tr_y, tr_nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(tr_wx, d_tr_wx, tr_nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(tr_wy, d_tr_wy, tr_nbytes, hipMemcpyDeviceToHost);
#endif /* TRAJECTORY */

    // stop
    printf("stop\n");
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    printf("time spent executing by the GPU: %.2f\n", gpu_time);

    checkCudaErrors(hipDeviceSynchronize());

    // normalize
    unsigned long long int sum = 0;

    for(unsigned long long int i = 0; i < nBins; ++i)
    {
        sum += uint64_t_concentration[i];
    }

    printf("sum = %lli\n", sum);

    normalizeArray(concentration, uint64_t_concentration,nBins);

    // save distribution
    printf("save dist\n");
    saveHist(concentration, argv[2], nBins);
    
#ifdef TRAJECTORY   
    saveHist(tr_x, argv[3], tr_points);
    saveHist(tr_y, argv[4], tr_points);
    saveHist(tr_wx, argv[5], tr_points);
    saveHist(tr_wy, argv[6], tr_points);
#endif /* TRAJECTORY */

    // free memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(concentration));
    checkCudaErrors(hipHostFree(uint64_t_concentration));
    checkCudaErrors(hipFree(d_concentration));
    checkCudaErrors(hipFree(devState));


#ifdef TRAJECTORY
    checkCudaErrors(hipHostFree(tr_x));
    checkCudaErrors(hipHostFree(tr_y));
    checkCudaErrors(hipHostFree(tr_wx));
    checkCudaErrors(hipHostFree(tr_wy));
    checkCudaErrors(hipFree(d_tr_x));
    checkCudaErrors(hipFree(d_tr_y));
    checkCudaErrors(hipFree(d_tr_wx));
    checkCudaErrors(hipFree(d_tr_wy));
#endif /* TRAJECTORY */

    return EXIT_SUCCESS;
}
